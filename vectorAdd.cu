#include "hip/hip_runtime.h"
/*----------
* Universidad del Valle
* Programación de Microprocesadores
*Mario Alejandro Sarmientos Inestroza 17055
*Fernando José Garavito Ovando 18071
*/
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x,string data)
{
  int one=two=zero=0;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  //if (i < n) y[i] = a*x[i] + y[i];
  for(int j = 0; i<=991353;i++){
    if(data[j] == "1"){
        one++;
    }
    if(data[j] == "2"){
        two++;
    }
    if(data[j] == "0"){
        zero++;
    }

  }
  
}
string intoF(string data){
    mod = (data.size()%2);
    while(mod != 0){
        temp = temp+"0";j
        mod = (temp.size()%4);
    }
}

int main(void)
{
  //Get data from txt files. 
  std::ifstream ifs("datos2.txt");
  std::string content( (std::istreambuf_iterator<char>(ifs) ),
                       (std::istreambuf_iterator<char>()    ) );

    

  //std::cout<<content<<endl;

  //padding a numero par. 
  intoF(content);

  int N = 1<<20;
  float *x,*d_x;
  x = (int)malloc(N*sizeof(int));


  hipMalloc(&d_x, N*sizeof(float)); 

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}